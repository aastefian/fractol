#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../includes/fractol.h"
#include "../libft/libft.h"
#include <stdio.h>
# define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
__global__ void md_2(unsigned int *a, unsigned int constw, unsigned int consth, float middlex, float middley, float scale, unsigned int max)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int index = row * constw + col;
  if(col >= constw || row >= consth) return;
	float cr = ((col - ((float)(DEF_WIN_WIDTH - 2) / 2)) / scale) - middlex;
	float ci = ((row - ((float)(DEF_WIN_HEIGHT - 2) / 2)) / scale) + middley;
    float zn_1r = 0;
    float zn_1i = 0;
    float zn_r = 0;
    float zn_i = 0;
   unsigned int iteration = 0;
  while (iteration < max)
  {
    zn_r = (zn_1r * zn_1r) - (zn_1i * zn_1i) + cr;
	zn_i = 2 * (zn_1r * zn_1i) + ci;
	zn_1r = zn_r;
	zn_1i = zn_i;
	if ((zn_r * zn_r + zn_i * zn_i) > 4)
		break;
	iteration++;
  }
  a[index] = iteration;
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		(void)file;
		(void)line;
		if (abort) exit(code);
	}
}

extern "C" void cuda_call_md(unsigned int *a_h, unsigned int constw, unsigned int consth, float middlex, float middley, float scale, unsigned int max, unsigned int reset)
{
  static unsigned int *a_d = NULL;
  static size_t size = 0;
  static dim3 block_size(16, 16);
  static dim3 grid_size(DEF_WIN_WIDTH / block_size.x + (DEF_WIN_WIDTH - DEF_WIN_WIDTH / block_size.x), DEF_WIN_HEIGHT / block_size.y + (DEF_WIN_HEIGHT - DEF_WIN_HEIGHT / block_size.y));  
  if (!reset)
  {
    if (size == 0)
    {
      size = constw * consth * sizeof(unsigned int);
      gpuErrchk(hipMalloc((void **) &a_d, size));
    }
   md_2 <<< grid_size, block_size, 0 >>> (a_d, constw, consth, middlex, middley, scale, max);
   gpuErrchk(hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost));
  }
  else
  {
    hipFree(a_d);
  }
}
