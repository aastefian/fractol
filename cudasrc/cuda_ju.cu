#include "hip/hip_runtime.h"
#include "../includes/fractol.h"
#include "../libft/libft.h"
# define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
__global__ void julia(unsigned int *a, unsigned int constw, unsigned int consth, float middlex, float middley, float zoom, t_com c, unsigned int max)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int index = row * constw + col;
	float cr;
	float ci;
  if(col >= constw || row >= consth) return;

	cr = c.r;
	ci = c.i;
	float zn_1r = (((float)(((col) - DEF_WIN_WIDTH / 2 + 1) * 2) / (float)((DEF_WIN_WIDTH - 2) / 2)) / zoom ) - middlex;
	float zn_1i = (((float)((DEF_WIN_HEIGHT / 2 - (row) - 1) * 2) / (float)((DEF_WIN_HEIGHT - 2) / 2)) / zoom ) - middley;
    float zn_r;
    float zn_i;
   unsigned int iteration = 0;
  while (iteration < max)
  {
    zn_r = (zn_1r * zn_1r) - (zn_1i * zn_1i) + cr;
	zn_i = 2 * zn_1r * zn_1i + ci;
	zn_1r = zn_r;
	zn_1i = zn_i;
	if ((zn_r * zn_r + zn_i * zn_i) > 4)
		break;
	iteration++;
  }
  a[index] = iteration;
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		(void)file;
		(void)line;
		if (abort) exit(code);
	}
}

extern "C" void cuda_call_julia(unsigned int *a_h, unsigned int constw, unsigned int consth, float middlex, float middley, float scale, t_com c, unsigned int max, unsigned int reset)
{
  static unsigned int *a_d = NULL;
  static size_t size = 0;
  static dim3 block_size(16, 16);
  static dim3 grid_size(DEF_WIN_WIDTH / block_size.x + (DEF_WIN_WIDTH - DEF_WIN_WIDTH / block_size.x), DEF_WIN_HEIGHT / block_size.y + (DEF_WIN_HEIGHT - DEF_WIN_HEIGHT / block_size.y));  
  if (!reset)
  {
    if (size == 0)
    {
      size = constw * consth * sizeof(unsigned int);
      gpuErrchk(hipMalloc((void **) &a_d, size));
    }
   julia<<< grid_size, block_size, 0 >>> (a_d, constw, consth, middlex, middley, scale, c, max);
   gpuErrchk(hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost));
  }
  else
  {
    hipFree(a_d);
  }
}
